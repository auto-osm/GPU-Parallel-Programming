#include "hip/hip_runtime.h"
#include <string.h>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
using namespace cv;

hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes);

int N;
int R;  //  rows 
int C;  // columns 
double ScaleFactor;
double CRA,SRA;
char *filename;

__global__ void lab5_kernel(uchar *GPU_i, uchar *GPU_o, int R, int C, double SRA, double CRA, double ScaleFactor)
{

	int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
	int j = blockIdx.y * blockDim.y + threadIdx.y;  // column of image
 
  double o_r, o_c; 
	int r,c;
  double nr,nc;
  nr = (double)R/2 - (double)i;
  nc = (double) j - (double)C/2;
	
	o_c = CRA*nc - SRA*nr ; // col
	o_r = SRA*nc + CRA*nr; //row
	o_c = o_c * ScaleFactor; 
	o_r = o_r * ScaleFactor;
 
  r = R/2 - (int)o_r ;
  c = (int)o_c + C/2;
   
	int odx = r*C*3 +   3*c;
	int idx = i*C*3 +   3*j; 
	
	  GPU_o[odx] = GPU_i[idx];
    GPU_o[odx+1] = GPU_i[idx+1];  // no change, REPLACE THIS
    GPU_o[odx+2] = GPU_i[idx+2]; 
    		
}


int main(int argc, char *argv[])
{
	float GPURuntimes[4];		
	hipError_t cudaStatus;
  int i ;

	if( argc != 4) {
	  printf("Usage: %s <input image> <output image> <numberOfimages>\n", argv[0]);
		exit(EXIT_FAILURE);
	}
 
	N = atoi(argv[3]);
 
 if(N< 1 || N > 30){
	  printf("Value of N %d should be less than 30 \n", argv[3]);
		exit(EXIT_FAILURE);
	}
 
 for (i = 1 ; i < N ; i ++ )
  
  {
 	
	int *CPU_OutputArray;
	double RotAngle;
  int RotDegrees;
  double Diagonal, H, V;
  
	Mat image;
    image = imread(argv[1], CV_LOAD_IMAGE_COLOR);
 
	if(!image.data) {
		fprintf(stderr, "Could not open or find the image.\n");
		exit(EXIT_FAILURE);
	}
	
	printf("Loaded image '%s', size = %dx%d (dims = %d).\n", argv[1], image.rows, image.cols, image.dims);
 
 	R = image.rows;
  C = image.cols;
  

      	CPU_OutputArray = (int*)malloc(3*C*R*sizeof(int));
      	if (CPU_OutputArray == NULL) {
      		fprintf(stderr, "OOPS. Can't create CPU_OutputArray using malloc() ...\n");
      		exit(EXIT_FAILURE);
      	}
      	
      	// Calculation for rotating an image
          int k; 
          
          for(k=0;k< 3*R*C; k++){
          CPU_OutputArray[k] = 0;
          }
        
        	RotDegrees = i *(360/N);
          printf("%d\n",	RotDegrees );
        	H=(double)C;
        	V=(double)R;
        	Diagonal=sqrt(H*H+V*V);
        	ScaleFactor =(C>R) ? V/Diagonal : H/Diagonal;
        	RotAngle=2*3.141592/360.000*(double)RotDegrees;
        	CRA=cos(RotAngle);
        	SRA=sin(RotAngle);
         
      	cudaStatus = launch_helper(image, CPU_OutputArray, GPURuntimes);
      	if (cudaStatus != hipSuccess) {
      		fprintf(stderr, "launch_helper failed!\n");
      		free(CPU_OutputArray);
      		exit(EXIT_FAILURE);
      	}
      
      	printf("-----------------------------------------------------------------\n");
      	printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \n Total=%5.2f ms\n",
      			GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
      	printf("-----------------------------------------------------------------\n");
      
      
      	cudaStatus = hipDeviceReset();
      	if (cudaStatus != hipSuccess) {
      		fprintf(stderr, "hipDeviceReset failed!\n");
      		free(CPU_OutputArray);
      		exit(EXIT_FAILURE);
      	}
      
      	Mat result = Mat(R, C, CV_8UC3, CPU_OutputArray);
        
        char * output_filename = argv[2];
        char fn[100]; 
       
        sprintf(fn,"dogR%03d.bmp",i);
        
      	if (!imwrite(fn, result)) {
      		fprintf(stderr, "couldn't write output to disk!\n");
      		free(CPU_OutputArray);
      		exit(EXIT_FAILURE);
      	}
       
      	printf("Saved image '%s', size = %dx%d (dims = %d).\n",
      	       fn, result.rows, result.cols, result.dims);
      
      	free(CPU_OutputArray);
 
 }
	exit(EXIT_SUCCESS);
}

hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes)
{
	hipEvent_t time1, time2, time3, time4; 
	int TotalGPUSize;
	uchar *GPU_idata;
	uchar *GPU_odata;

	dim3 threadsPerBlock;
	dim3 numBlocks;

	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);  
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

	hipEventRecord(time1, 0);

    TotalGPUSize = 3*R*C*sizeof(uchar);
 
	cudaStatus = hipMalloc((void**)&GPU_idata, TotalGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
 
	cudaStatus = hipMalloc((void**)&GPU_odata, TotalGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	cudaStatus = hipMemcpy(GPU_idata, image.data, TotalGPUSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}
 
   cudaStatus = hipMemcpy(GPU_odata,CPU_OutputArray,3*R*C, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy failed!\n");
    goto Error;
  }


	hipEventRecord(time2, 0);

	threadsPerBlock = dim3(8, 8);
	numBlocks = dim3(R/ threadsPerBlock.y, C/ threadsPerBlock.x);
	lab5_kernel<<<numBlocks, threadsPerBlock>>>(GPU_idata, GPU_odata, R, C, SRA, CRA, ScaleFactor);


	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "error code %d (%s) launching kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipEventRecord(time3, 0);

	cudaStatus = hipMemcpy(CPU_OutputArray, GPU_odata, TotalGPUSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	hipEventRecord(time4, 0);
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;

	Error:
	hipFree(GPU_odata);
	hipFree(GPU_idata);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);

	return cudaStatus;
}
