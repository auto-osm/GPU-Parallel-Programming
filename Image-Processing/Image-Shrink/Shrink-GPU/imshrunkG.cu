#include "hip/hip_runtime.h"
// ECE 406 Lab 5, Fall 2015

#include <stdio.h>

// CUDA stuff:
#include "hip/hip_runtime.h"
#include ""

// OpenCV stuff (note: C++ not C):
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
using namespace cv;

hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes);

int M;  // number of rows in image
int N;  // number of columns in image

unsigned int shrinkRatiox,shrinkRatioy;
unsigned int x,y;

// These come from CLI arguments:

__global__ void lab5_kernel(uchar *GPU_i, uchar *GPU_o, int M, int N, int x, int y, int shrinkRatiox, int shrinkRatioy)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;  // row of image
	int j = blockIdx.y * blockDim.y + threadIdx.y;  // column of image
	int odx = i*x*3 +   3*j;  // which pixel in full 1D array
	int idx=i*shrinkRatioy*N*3+3*j*shrinkRatiox;
  if(odx<x*y*3 && idx<M*N*3) {
  
	GPU_o[odx] = GPU_i[idx];
    GPU_o[odx+1] = GPU_i[idx+1];  // no change, REPLACE THIS
    GPU_o[odx+2] = GPU_i[idx+2];  // no change, REPLACE THIS
  }
}

int main(int argc, char *argv[])
{
	float GPURuntimes[4];		// run times of the GPU code
	hipError_t cudaStatus;
	int *CPU_OutputArray;		// where the GPU should copy the output back to

	if( argc != 5) {
	  printf("Usage: %s <input image> <output image> <shrinkRatiox> <shrinkRatioy>\n", argv[0]);
		exit(EXIT_FAILURE);
	}
	shrinkRatiox=atoi(argv[3]);
	shrinkRatioy=atoi(argv[4]);

	Mat image;
  image = imread(argv[1], CV_LOAD_IMAGE_COLOR);
 
	// we could load it as CV_LOAD_IMAGE_COLOR, but we don't want to worry about that extra dimension
	if(! image.data ) {
		fprintf(stderr, "Could not open or find the image.\n");
		exit(EXIT_FAILURE);
	}
	printf("Loaded image '%s', size = %dx%d (dims = %d).\n", argv[1], image.rows, image.cols, image.dims);

	// Set up global variables based on image size:
	M = image.rows;
	N = image.cols;
	
	x=(N/shrinkRatiox);
	y=(M/shrinkRatioy);
  
  printf("\n%d %d %d %d\n", x,y,M,N);
  

	CPU_OutputArray = (int*)malloc(3*x*y*sizeof(int));
  if (CPU_OutputArray == NULL) {
		fprintf(stderr, "OOPS. Can't create CPU_OutputArray using malloc() ...\n");
		exit(EXIT_FAILURE);
	}

	// Run it:
	cudaStatus = launch_helper(image, CPU_OutputArray, GPURuntimes);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "launch_helper failed!\n");
		free(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}

	printf("-----------------------------------------------------------------\n");
	printf("Tfr CPU->GPU = %5.2f ms ... \nExecution = %5.2f ms ... \nTfr GPU->CPU = %5.2f ms   \n Total=%5.2f ms\n",
			GPURuntimes[1], GPURuntimes[2], GPURuntimes[3], GPURuntimes[0]);
	printf("-----------------------------------------------------------------\n");

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Parallel Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!\n");
		free(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}

	// Display the output image:
	//Mat result = Mat(y, x, CV_8UC1, CPU_OutputArray);
	Mat result = Mat(y, x, CV_8UC3, CPU_OutputArray);
  // and save it to disk:
	string output_filename = argv[2];
	if (!imwrite(output_filename, result)) {
		fprintf(stderr, "couldn't write output to disk!\n");
		free(CPU_OutputArray);
		exit(EXIT_FAILURE);
	}
	printf("Saved image '%s', size = %dx%d (dims = %d).\n",
	       output_filename.c_str(), result.rows, result.cols, result.dims);

	free(CPU_OutputArray);
	exit(EXIT_SUCCESS);
}

// Helper function for launching a CUDA kernel (including memcpy, timing, etc.):
hipError_t launch_helper(Mat image, int *CPU_OutputArray, float* Runtimes)
{
	hipEvent_t time1, time2, time3, time4;
	int TotalOutputGPUSize;  // total size of 1 image (i.e. input or output) in bytes
	int TotalGPUSize;
	uchar *GPU_idata;
	uchar *GPU_odata;
	// Note that we could store GPU_i and GPU_o as 2D arrays instead of 1D...
	// it would make indexing simpler, but could complicate memcpy.

	dim3 threadsPerBlock;
	dim3 numBlocks;

	// Choose which GPU to run on; change this on a multi-GPU system.
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);  // use the first GPU (not necessarily the fastest)
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n");
		goto Error;
	}

	hipEventCreate(&time1);
	hipEventCreate(&time2);
	hipEventCreate(&time3);
	hipEventCreate(&time4);

	hipEventRecord(time1, 0);

  
	TotalGPUSize = 3*M * N * sizeof(uchar);
   TotalOutputGPUSize = 3*x * y * sizeof(uchar);
 
	cudaStatus = hipMalloc((void**)&GPU_idata, TotalGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}
 
	cudaStatus = hipMalloc((void**)&GPU_odata, TotalOutputGPUSize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!\n");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(GPU_idata, image.data, TotalGPUSize, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	hipEventRecord(time2, 0);

	// Launch a kernel on the GPU with one thread for each pixel.
	threadsPerBlock = dim3(8, 8);
	numBlocks = dim3(y / threadsPerBlock.y, x / threadsPerBlock.x);
	lab5_kernel<<<numBlocks, threadsPerBlock>>>(GPU_idata, GPU_odata, M, N, x, y, shrinkRatiox, shrinkRatioy);

	// Check for errors immediately after kernel launch.
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "error code %d (%s) launching kernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d (%s) after launching addKernel!\n", cudaStatus, hipGetErrorString(cudaStatus));
		goto Error;
	}

	hipEventRecord(time3, 0);

	// Copy output (results) from GPU buffer to host (CPU) memory.
	cudaStatus = hipMemcpy(CPU_OutputArray, GPU_odata, TotalOutputGPUSize, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!\n");
		goto Error;
	}

	hipEventRecord(time4, 0);
	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);

	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime;

	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	Runtimes[0] = totalTime;
	Runtimes[1] = tfrCPUtoGPU;
	Runtimes[2] = kernelExecutionTime;
	Runtimes[3] = tfrGPUtoCPU;

	Error:
	hipFree(GPU_odata);
	hipFree(GPU_idata);
	hipEventDestroy(time1);
	hipEventDestroy(time2);
	hipEventDestroy(time3);
	hipEventDestroy(time4);

	return cudaStatus;
}
